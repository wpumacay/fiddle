
#include <PSOcudaHelpers.h>

using namespace std;

namespace optimization
{


    void cuPSOcreateParticles( PSOcudaParticlesInfo& devParticlesInfo )
    {
        hipError_t _errorCode;

        _errorCode = hipMalloc( ( void** ) &devParticlesInfo.pos, devParticlesInfo.vecBufferSize );
        cout << "errCode: " << _errorCode << endl;

        _errorCode = hipMalloc( ( void** ) &devParticlesInfo.vel, devParticlesInfo.vecBufferSize );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMalloc( ( void** ) &devParticlesInfo.bpos, devParticlesInfo.vecBufferSize );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMalloc( ( void** ) &devParticlesInfo.cost, devParticlesInfo.costBufferSize );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMalloc( ( void** ) &devParticlesInfo.bcost, devParticlesInfo.costBufferSize );
        cout << "errCode: " << _errorCode << endl;        
    }

    void cuPSOinitParticles( PSOcudaParticlesInfo& hostParticlesInfo,
                             PSOcudaParticlesInfo& devParticlesInfo,
                             double domainMin, double domainMax, BaseFunction* fcn )
    {
        // Initialize host particles to be sent to device
        for ( int p = 0; p < hostParticlesInfo.population; p++ )
        {
            for ( int d = 0; d < hostParticlesInfo.ndim; d++ )
            {
                hostParticlesInfo.pos[ p * hostParticlesInfo.ndim + d ] = RANDOM( domainMin, domainMax );
                hostParticlesInfo.vel[ p * hostParticlesInfo.ndim + d ] = 0.0;
                hostParticlesInfo.bpos[ p * hostParticlesInfo.ndim + d ] = hostParticlesInfo.pos[ p * hostParticlesInfo.ndim + d ];
            }

            hostParticlesInfo.cost[ p ]  = fcn->eval( hostParticlesInfo.pos + p * hostParticlesInfo.ndim, d );
            hostParticlesInfo.bcost[ p ] = hostParticlesInfo.cost[ p ];
        }

        // Pass this info to the device particles
        hipError_t _errorCode;

        _errorCode = hipMemcpy( devParticlesInfo.pos, hostParticlesInfo.pos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMemcpy( devParticlesInfo.vel, hostParticlesInfo.vel, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMemcpy( devParticlesInfo.bpos, hostParticlesInfo.bpos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMemcpy( devParticlesInfo.cost, hostParticlesInfo.cost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyHostToDevice );
        cout << "errCode: " << _errorCode << endl;        

        _errorCode = hipMemcpy( devParticlesInfo.bcost, hostParticlesInfo.bcost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyHostToDevice );
        cout << "errCode: " << _errorCode << endl;        
    }

    void cuPSOupdateParticles( PSOcudaParticlesInfo& hostParticlesInfo,
                               PSOcudaParticlesInfo& devParticlesInfo )
    {
        // Copy from host to device


        // Launch kernels


        // Copy back from device to host



    }

    void cuPSOreleaseParticles( PSOcudaParticlesInfo& devParticlesInfo )
    {

    }




}