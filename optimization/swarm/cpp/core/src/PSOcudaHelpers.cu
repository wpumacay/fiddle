#include "hip/hip_runtime.h"

#include <PSOcudaHelpers.h>

using namespace std;

#define MAX_THREADS_PER_BLOCK 1024


__device__ double DF_FunctionSphere( double* d_ppos, int ndim, int pIndx )
{
    double _res = 0.0;

    for ( int d = 0; d < ndim; d++ )
    {
        double _xd = d_ppos[ pIndx * ndim + d ];

        _res += _xd * _xd;
    }

    return _res;
}

__device__ double DF_FunctionAckley( double* d_ppos, int ndim, int pIndx )
{
    // a = 20, b = 0.2, c = 2.0 * pi

    double _sumsquares = 0.0;
    for ( int d = 0; d < ndim; d++ )
    {
        double _xd = d_ppos[ pIndx * ndim + d ];

        _sumsquares += _xd * _xd;
    }
    double _t1 = -20.0 * exp( -0.2 * sqrt( _sumsquares / ndim ) );

    double _sumcos = 0.0;
    for ( int d = 0; d < ndim; d++ )
    {
        _sumcos += cos( 2.0 * 3.141592653589793 * d_ppos[ pIndx * ndim + d ] );
    }
    double _t2 = -exp( _sumcos / ndim );
    double _t3 = 20.0 + 2.718281828459045;

    return _t1 + _t2 + _t3;
}

__device__ double DF_FunctionSchwefel( double* d_ppos, int ndim, int pIndx )
{
    double _res = 418.9829 * ndim;

    for ( int d = 0; d < ndim; d++ )
    {
        double _xd = d_ppos[ pIndx * ndim + d ];

        _res -= _xd * sin( sqrt( fabs( _xd ) ) );
    }

    return _res;
}

__device__ double DF_FunctionSchafferFcn6( double* d_ppos, int ndim, int pIndx )
{
    // a = 20, b = 0.2, c = 2.0 * pi

    double _sumsquares = 0.0;
    for ( int d = 0; d < ndim; d++ )
    {
        double _xd = d_ppos[ pIndx * ndim + d ];

        _sumsquares += _xd * _xd;
    }

    double _tmp1 = sin( sqrt( _sumsquares ) );
    double _numerator = _tmp1 * _tmp1 - 0.5;

    double _tmp2 = 1.0 + 0.001 * _sumsquares;
    double _denominator = _tmp2 * _tmp2;

    return 0.5 - _numerator / _denominator;
}

__global__ void kernel_initializeRandomGenerator( unsigned int seed, int populationSize, hiprandState_t* states )
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if ( id > populationSize - 1 )
    {
        // Out of bounds of population buffer, just exit
        return;
    }

    hiprand_init( seed, id, 0, &states[id] );
}


__global__ void kernel_updateParticles( double* d_ppos, double* d_pvel, double* d_pbpos,
                                        double* d_pcost, double* d_pbcost, 
                                        double* d_gbestpos, double d_gbestcost,
                                        int ndim, int populationSize,
                                        double domainMin, double domainMax,
                                        double vmin, double vmax,
                                        double w, double c1, double c2, double k,
                                        int fcnObjId, bool isMinimization,
                                        hiprandState_t* randomStates )
{

    int p = threadIdx.x + blockIdx.x * blockDim.x;

    if ( p > populationSize - 1 )
    {
        // Out of bounds of population buffer, just exit
        return;
    }

    // Update velocity of the particle *******************************
    double _vnorm = 0.0;

    for ( int d = 0; d < ndim; d++ )
    {
        double _v_d = w * d_pvel[ p * ndim + d ] + 
                      c1 * hiprand_uniform( &randomStates[p] ) * ( d_pbpos[ p * ndim + d ] - d_ppos[ p * ndim + d ] ) +
                      c2 * hiprand_uniform( &randomStates[p] ) * ( d_gbestpos[ d ] - d_ppos[ p * ndim + d ] );

        d_pvel[ p * ndim + d ] = _v_d;
        _vnorm += _v_d * _v_d;
    }

    _vnorm = sqrt( _vnorm );
    if ( _vnorm > 0.001 )
    {
        double _vClamped = ( _vnorm < vmin ) ? ( vmin ) : ( ( _vnorm > vmax ) ? ( vmax ) : ( _vnorm ) );

        for ( int d = 0; d < ndim; d++ )
        {
            d_pvel[ p * ndim + d ] = ( d_pvel[ p * ndim + d ] / _vnorm ) * _vClamped;
        }
    }

    // Update position of the particle *******************************

    for ( int d = 0; d < ndim; d++ )
    {
        double _x_d = d_ppos[ p * ndim + d ];

        _x_d += d_pvel[ p * ndim + d ];

        _x_d = ( _x_d < domainMin ) ? ( domainMin ) : ( ( _x_d > domainMax ) ? ( domainMax ) : ( _x_d ) );

        d_ppos[ p * ndim + d ] = _x_d;
    }

    // Update cost of the particle ***********************************

    if ( fcnObjId == OPT_FUNCTION_SPHERE )
    {
        d_pcost[ p ] = DF_FunctionSphere( d_ppos, ndim, p );
    }
    else if ( fcnObjId == OPT_FUNCTION_ACKLEY )
    {
        d_pcost[ p ] = DF_FunctionAckley( d_ppos, ndim, p );
    }
    else if ( fcnObjId == OPT_FUNCTION_SCHWEFEL )
    {
        d_pcost[ p ] = DF_FunctionSchwefel( d_ppos, ndim, p );
    }
    else if ( fcnObjId == OPT_FUNCTION_SCHAFFER6 )
    {
        d_pcost[ p ] = DF_FunctionSchafferFcn6( d_ppos, ndim, p );
    }

    if ( ( isMinimization && ( d_pcost[ p ] < d_pbcost[ p ] ) ) ||
         ( !isMinimization && ( d_pcost[ p ] > d_pbcost[ p ] ) ) )
    {
        d_pbcost[ p ] = d_pcost[ p ];
        for ( int d = 0; d < ndim; d++ ) 
        { 
            d_pbpos[ p * ndim + d ] = d_ppos[ p * ndim + d ]; 
        }
    }
}






namespace optimization
{


    void cuPSOcreateParticles( PSOcudaParticlesInfo& devParticlesInfo )
    {
        // hipError_t _errorCode;

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.pos, devParticlesInfo.vecBufferSize );
        // cout << "errCode - pos: " << _errorCode << endl;

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.vel, devParticlesInfo.vecBufferSize );
        // cout << "errCode - vel: " << _errorCode << endl;        

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.bpos, devParticlesInfo.vecBufferSize );
        // cout << "errCode - bpos: " << _errorCode << endl;        

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.cost, devParticlesInfo.costBufferSize );
        // cout << "errCode - cost: " << _errorCode << endl;        

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.bcost, devParticlesInfo.costBufferSize );
        // cout << "errCode - bcost: " << _errorCode << endl;        

        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.gbestpos, devParticlesInfo.vecSize );
        // cout << "errCode - gbestpos: " << _errorCode << endl;        

        // Initialize random number generator as well
        /*_errorCode = */hipMalloc( ( void** ) &devParticlesInfo.devRandStates, sizeof( hiprandState_t ) * devParticlesInfo.population );
        // cout << "errCode - devRandStates: " << _errorCode << endl;        

        int _bx = ceil( ( ( float ) devParticlesInfo.population ) / MAX_THREADS_PER_BLOCK );

        dim3 _blockSize( MAX_THREADS_PER_BLOCK );
        dim3 _gridSize( _bx );

        kernel_initializeRandomGenerator<<< _gridSize, _blockSize >>>( time( 0 ), devParticlesInfo.population, devParticlesInfo.devRandStates );
    }

    void cuPSOinitParticles( PSOcudaParticlesInfo& hostParticlesInfo,
                             PSOcudaParticlesInfo& devParticlesInfo, 
                             BaseFunction* fcn, bool isMinimization )
    {

        hostParticlesInfo.gbestcost = ( isMinimization ) ? 1000000.0 : -1000000.0;

        // Initialize host particles to be sent to device
        for ( int p = 0; p < hostParticlesInfo.population; p++ )
        {
            for ( int d = 0; d < hostParticlesInfo.ndim; d++ )
            {
                hostParticlesInfo.pos[ p * hostParticlesInfo.ndim + d ] = RANDOM( hostParticlesInfo.domainMin, hostParticlesInfo.domainMax );
                hostParticlesInfo.vel[ p * hostParticlesInfo.ndim + d ] = 0.0;
                hostParticlesInfo.bpos[ p * hostParticlesInfo.ndim + d ] = hostParticlesInfo.pos[ p * hostParticlesInfo.ndim + d ];
            }

            hostParticlesInfo.cost[ p ]  = fcn->eval( hostParticlesInfo.pos + p * hostParticlesInfo.ndim, hostParticlesInfo.ndim );
            hostParticlesInfo.bcost[ p ] = hostParticlesInfo.cost[ p ];

            if ( ( isMinimization && ( hostParticlesInfo.cost[ p ] < hostParticlesInfo.gbestcost ) ) ||
                 ( !isMinimization && ( hostParticlesInfo.cost[ p ] > hostParticlesInfo.gbestcost ) ) )
            {
                hostParticlesInfo.gbestcost = hostParticlesInfo.cost[ p ];
                for ( int d = 0; d < hostParticlesInfo.ndim; d++ )
                {
                    hostParticlesInfo.gbestpos[ d ] = hostParticlesInfo.pos[ p * hostParticlesInfo.ndim + d ];
                }
            }
        }

        // Pass this info to the device particles
        devParticlesInfo.gbestcost = ( isMinimization ) ? 1000000.0 : -1000000.0;

        // hipError_t _errorCode;

        /*_errorCode = */hipMemcpy( devParticlesInfo.pos, hostParticlesInfo.pos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        // cout << "errCode pos: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( devParticlesInfo.vel, hostParticlesInfo.vel, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        // cout << "errCode vel: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( devParticlesInfo.bpos, hostParticlesInfo.bpos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyHostToDevice );
        // cout << "errCode bpos: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( devParticlesInfo.cost, hostParticlesInfo.cost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyHostToDevice );
        // cout << "errCode cost: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( devParticlesInfo.bcost, hostParticlesInfo.bcost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyHostToDevice );
        // cout << "errCode bcost: " << _errorCode << endl;        
    }

    void cuPSOupdateParticles( PSOcudaParticlesInfo& hostParticlesInfo,
                               PSOcudaParticlesInfo& devParticlesInfo,
                               double w, double c1, double c2, double k,
                               int fcnObjId, bool isMinimization )
    {
        // Copy from host to device - only best cost and best pos
        // hipError_t _errorCode;

        /*_errorCode = */hipMemcpy( devParticlesInfo.gbestpos, hostParticlesInfo.gbestpos, 
                                 hostParticlesInfo.vecSize, hipMemcpyHostToDevice );
        // cout << "errCode gbestpos: " << _errorCode << endl;       

        devParticlesInfo.gbestcost = hostParticlesInfo.gbestcost;// this one is just a copy in host

        // Launch kernels ******************************************************

        int _bx = ceil( ( ( float ) devParticlesInfo.population ) / MAX_THREADS_PER_BLOCK );

        dim3 _blockSize( MAX_THREADS_PER_BLOCK );
        dim3 _gridSize( _bx );

        kernel_updateParticles<<< _gridSize, _blockSize >>>( devParticlesInfo.pos, devParticlesInfo.vel, devParticlesInfo.bpos,
                                                             devParticlesInfo.cost, devParticlesInfo.bcost,
                                                             devParticlesInfo.gbestpos, devParticlesInfo.gbestcost,
                                                             devParticlesInfo.ndim, devParticlesInfo.population,
                                                             devParticlesInfo.domainMin, devParticlesInfo.domainMax,
                                                             devParticlesInfo.vmin, devParticlesInfo.vmax,
                                                             w, c1, c2, k,
                                                             fcnObjId, isMinimization,
                                                             devParticlesInfo.devRandStates );

        // *********************************************************************


        // Copy back from device to host ***************************************

        /*_errorCode = */hipMemcpy( hostParticlesInfo.pos, devParticlesInfo.pos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyDeviceToHost );
        // cout << "errCode--pos: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( hostParticlesInfo.vel, devParticlesInfo.vel, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyDeviceToHost );
        // cout << "errCode--vel: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( hostParticlesInfo.bpos, devParticlesInfo.bpos, 
                                 hostParticlesInfo.vecBufferSize, hipMemcpyDeviceToHost );
        // cout << "errCode--bpos: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( hostParticlesInfo.cost, devParticlesInfo.cost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyDeviceToHost );
        // cout << "errCode--cost: " << _errorCode << endl;        

        /*_errorCode = */hipMemcpy( hostParticlesInfo.bcost, devParticlesInfo.bcost, 
                                 hostParticlesInfo.costBufferSize, hipMemcpyDeviceToHost );
        // cout << "errCode--bcost: " << _errorCode << endl;       

        // *********************************************************************

        // Do check for new best in here - host ********************************

        int _bestParticleIndx = -1;

        for ( int p = 0; p < hostParticlesInfo.population; p++ )
        {
            if ( ( isMinimization && ( hostParticlesInfo.bcost[ p ] < hostParticlesInfo.gbestcost ) ) ||
                 ( !isMinimization && ( hostParticlesInfo.bcost[ p ] > hostParticlesInfo.gbestcost ) ) )
            {
                hostParticlesInfo.gbestcost = hostParticlesInfo.bcost[ p ];
                _bestParticleIndx = p;
            }
        }

        if ( _bestParticleIndx != -1 )
        {
            for ( int d = 0; d < hostParticlesInfo.ndim; d++ )
            {
                hostParticlesInfo.gbestpos[ d ] = hostParticlesInfo.bpos[ _bestParticleIndx * hostParticlesInfo.ndim + d ];
            }
        }
    }

    void cuPSOreleaseParticles( PSOcudaParticlesInfo& devParticlesInfo )
    {
        hipFree( devParticlesInfo.pos );
        hipFree( devParticlesInfo.vel );
        hipFree( devParticlesInfo.bpos );
        hipFree( devParticlesInfo.cost );
        hipFree( devParticlesInfo.bcost );
        hipFree( devParticlesInfo.gbestpos );
        hipFree( devParticlesInfo.devRandStates );
    }




}